#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define random(a, b) ((a) + rand() % ((b) - (a) + 1))
#define INF 0x3f3f3f3f3f3f3f3f
#define BLOCK_SIZE 256
#define CHECK(call)                                                           \
    {                                                                         \
        const hipError_t error = call;                                       \
        if (error != hipSuccess) {                                           \
            printf("Error: %s:%d, ", __FILE__, __LINE__);                     \
            printf("code:%d, reson: %s\n", error, hipGetErrorString(error)); \
            exit(-10 * error);                                                \
        }                                                                     \
    }
__host__ void initialize(int h_num_node, int *h_arc, int *h_path_node, int *h_shortLenTable) {
    srand(time(0));
    int idx = 0;
    // 数据输入 并分配内存

    // for (int i = 0; i < h_num_node; ++i) {
    //     for (int j = 0; j < h_num_node; ++j) {
    //         if (i == j)
    //             h_shortLenTable[idx] = h_arc[idx] = 0;
    //         else {
    //             h_arc[idx] = random(1, 1000) / 100.0;
    //             h_shortLenTable[idx] = h_arc[idx];
    //         }
    //     }
    // }
    int Prange = 4;
    for (int i = 0; i < h_num_node; i++) {
        for (int j = 0; j < h_num_node; j++) {
            if (i == j)
                h_shortLenTable[idx] = h_arc[idx] = 0;
            else {
                int pr = rand() % Prange;
                h_shortLenTable[idx] = h_arc[idx] = pr == 0 ? ((rand() % 997) + 1) : INF; //set edge random edge weight to random value, or to INF
            }
            h_path_node[idx] = -1;
            ++idx;
        }
    }
}
__host__ void hostCalShortTable(int h_num_node, int *h_arc, int *h_path_node, int *h_shortLenTable) {
    struct timeval start, end;
    gettimeofday(&start, NULL);
    int idx, idb, ida;
    for (int k = 0; k < h_num_node; ++k) {
        //第k个点的松弛
        for (int i = 0; i < h_num_node; ++i) {
            for (int j = 0; j < h_num_node; ++j) {
                idx = i * h_num_node + j;
                idb = i * h_num_node + k;
                ida = k * h_num_node + j;
                if (h_shortLenTable[idx] >
                    h_shortLenTable[idb] + h_shortLenTable[ida]) {
                    h_shortLenTable[idx] =
                        h_shortLenTable[idb] + h_shortLenTable[ida];
                    h_path_node[idx] = k;
                }
            }
        }
    }
    gettimeofday(&end, NULL);
    printf("CPU Time: %.6lfs\n", end.tv_usec / 1000000.0 + end.tv_sec -
                                     start.tv_usec / 1000000.0 - start.tv_sec);
}
__host__ bool checkResult(int h_num_node, int *h_path_node, int *h_shortLenTable, int *res_path_node, int *res_shortLenTable) {
    int tot = h_num_node * h_num_node;
    int idx = 0, idy = 0;
    for (int i = 0; i < tot; ++i) {
        if (h_path_node[i] != res_path_node[i] || h_shortLenTable[i] != res_shortLenTable[i]) {
            printf("Wrong Answer on %d to %d.\n", idx, idy);
            printf("Host   result: path_node[%d][%d] = %d, shortLenTable[%d][%d] = %.2f.\n", idx, idy, h_path_node[i], idx, idy, h_shortLenTable[i]);
            printf("Device result: path_node[%d][%d] = %d, shortLenTable[%d][%d] = %.2f.\n", idx, idy, res_path_node[i], idx, idy, res_shortLenTable[i]);
            return false;
        }
        ++idx;
        if (idx == h_num_node)
            idx = 0, ++idy;
    }
    puts("The results of GPU and CPU are the same.");
    return true;
}
__global__ void deviceCalBetter(int k, int d_num_node, int *d_path_node, int *d_shortLenTable) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int N = d_num_node;
    if (col >= N)
        return;
    int idx = N * blockIdx.y + col;

    __shared__ int best;
    if (threadIdx.x == 0)
        best = d_shortLenTable[N * blockIdx.y + k];
    __syncthreads();
    if (best == INF)
        return;
    int tmp_b = d_shortLenTable[k * N + col];
    if (tmp_b == INF)
        return;
    int cur = best + tmp_b;
    if (cur < d_shortLenTable[idx]) {
        d_shortLenTable[idx] = cur;
        d_path_node[idx] = k;
    }
}
__global__ void deviceCalShortTable(int k, int *d_num_node, int *d_path_node, int *d_shortLenTable) {
    unsigned int IDX = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ int N;
    N = *d_num_node;
    if (IDX >= N * N)
        return;
    //cal the i and j
    int i = IDX % N;
    int j = IDX / N;
    int idx = i * N + j;
    int idb = i * N + k;
    int ida = k * N + j;
    if (d_shortLenTable[idx] >
        d_shortLenTable[idb] + d_shortLenTable[ida]) {
        d_shortLenTable[idx] =
            d_shortLenTable[idb] + d_shortLenTable[ida];
        d_path_node[idx] = k;
    }
}
void shortestPath_floyd(int h_num_node, int *h_arc, int *h_path_node, int *h_shortLenTable) {
    int totIntBytes = h_num_node * h_num_node * sizeof(int *);
    int totintBytes = h_num_node * h_num_node * sizeof(int *);
    //设备端变量定义
    int d_num_node = h_num_node;
    int *d_arc;
    int *d_path_node;
    int *d_shortLenTable;

    hipMalloc((void **)&d_arc, totintBytes);
    hipMalloc((void **)&d_path_node, totIntBytes);
    hipMalloc((void **)&d_shortLenTable, totintBytes);

    //Host To Device
    hipMemcpy(d_arc, h_arc, totintBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_path_node, h_path_node, totIntBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_shortLenTable, h_shortLenTable, totintBytes, hipMemcpyHostToDevice);
    // printf("%d %d\n", d_num_node, h_num_node);

    //git
    dim3 dimGrid((h_num_node + BLOCK_SIZE - 1) / BLOCK_SIZE, h_num_node);
    for (int k = 0; k < h_num_node; ++k) {
        deviceCalBetter<<<dimGrid, BLOCK_SIZE>>>(k, d_num_node, d_path_node, d_shortLenTable);
    }
    hipDeviceSynchronize();

    // //分配任务
    // int calSize = h_num_node * h_num_node;
    // dim3 grid((calSize + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // for (int k = 0; k < h_num_node; ++k) {
    //     deviceCalShortTable<<<grid, BLOCK_SIZE>>>(k, d_num_node, d_path_node, d_shortLenTable);
    // }
    // cudaDeviceSynchronize();

    //record and check the result
    // int *res_path_node;
    // int *res_shortLenTable;
    // res_path_node = (int *)malloc(totIntBytes);
    // res_shortLenTable = (int *)malloc(totintBytes);
    // cudaMemcpy(res_path_node, d_path_node, totIntBytes, cudaMemcpyDeviceToHost);
    // cudaMemcpy(res_shortLenTable, d_shortLenTable, totintBytes, cudaMemcpyDeviceToHost);
    // hostCalShortTable(h_num_node, h_arc, h_path_node, h_shortLenTable);
    // checkResult(h_num_node, h_path_node, h_shortLenTable, res_path_node, res_shortLenTable);
    // free(res_path_node);
    // free(res_shortLenTable);

    hipFree(d_arc);
    hipFree(d_path_node);
    hipFree(d_shortLenTable);
}
__global__ void warmingup(int n) {
    unsigned int IDX = blockDim.x * blockIdx.x + threadIdx.x;
    if (IDX >= n)
        return;
}

int main(int argc, char **argv) {
    printf("%s Starting...\n", argv[0]);

    //get device information
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    //读文件
    //主机变量定义
    int h_num_node;       // 节点个数 70~50000
    int *h_arc;           // 邻接矩阵
    int *h_path_node;     // 最短路径
    int *h_shortLenTable; // 最短路径长度

    h_num_node = 1000;
    int totIntBytes = h_num_node * h_num_node * sizeof(int *);
    int totintBytes = h_num_node * h_num_node * sizeof(int *);
    h_arc = (int *)malloc(totintBytes);
    h_path_node = (int *)malloc(totIntBytes);
    h_shortLenTable = (int *)malloc(totintBytes);

    initialize(h_num_node, h_arc, h_path_node, h_shortLenTable);

    int calSize = h_num_node * h_num_node;
    dim3 grid((calSize + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 block(BLOCK_SIZE);
    warmingup<<<grid, block>>>(32);
    //初始化时间参数
    struct timeval start, end;
    gettimeofday(&start, NULL);
    //calculate on GPU
    shortestPath_floyd(h_num_node, h_arc, h_path_node, h_shortLenTable);
    //输出用时
    gettimeofday(&end, NULL);
    printf("GPU Time: %.6lfs\n", end.tv_usec / 1000000.0 + end.tv_sec -
                                     start.tv_usec / 1000000.0 - start.tv_sec);

    // hostCalShortTable(h_num_node, h_arc, h_path_node, h_shortLenTable);

    //释放内存
    free(h_arc);
    free(h_path_node);
    free(h_shortLenTable);

    hipDeviceReset();
    return 0;
}